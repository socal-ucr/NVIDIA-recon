#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

using namespace std;

const int GPUs[] = {0,1,2,3,4}; // If left blank all available GPUs will be used.

vector<int> g(GPUs, GPUs + sizeof(GPUs)/sizeof(int));

void configure(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d)
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipMalloc(&buffer_s[i], size);
        hipMalloc(&buffer_d[i], size);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceEnablePeerAccess(g[j], 0);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceEnablePeerAccess(g[i], 0);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void reset(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d)
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipFree(buffer_s[i]);
        hipFree(buffer_d[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceDisablePeerAccess(g[j]);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceDisablePeerAccess(g[i]);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void blocked_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d)
{
    float time_taken[g.size()*g.size()], bw[g.size()*g.size()];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("\nBlocked Memory Transfers: Only one memory transfer at a time\n");

    for (int i=0; i<g.size(); i++)
    {
        for (int j=0; j<g.size(); j++)
        {
            if (i!=j)
            {
                printf("Copying from %d to %d\n", g[i], g[j]);
                hipEventRecord(start);
                hipMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[j],g[j], size);
                hipEventRecord(stop);
                hipDeviceSynchronize();
                hipEventSynchronize(stop);
                float time_ms=0.0;
                hipEventElapsedTime(&time_ms, start, stop);
                time_taken[i*g.size()+j] = time_ms*1e3;
                bw[i*g.size()+j] = (float)size*1000/time_ms/(1<<30);
            }
        }
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Time(ms) spent in memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        {
            if (i==j)
                printf("%12.4f", 0.0);
            else
                printf("%12.4f", time_taken[i*g.size()+j]);
        }
        printf("\n");
    }

    printf("bandwidth(Gbps) utilized during memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        if (i==j)
            printf("%12.4f", 0.0);
        else
            printf("%12.4f", bw[i*g.size()+j]);
        printf("\n");
    }
}

void perf_analyze(size_t size)
{
    vector<int*> buffer_s(g.size());
    vector<int*> buffer_d(g.size());

    configure(size, buffer_s, buffer_d);

    // Cyclic
    blocked_copy(size, buffer_s, buffer_d);

    reset(size, buffer_s, buffer_d);
}

int main(int argc, char** argv)
{
    // NVLink D<->D performance
    size_t size = (1<<30);
    if (!g.size())
    {
        int n;
        printf("Using all 8 GPUs\n");
        hipGetDeviceCount(&n);
        for (int i=0; i<n; i++)
            g.push_back(i);
    }
    //define size
    perf_analyze(size);

    return 0;
}
