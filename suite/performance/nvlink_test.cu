#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

using namespace std;

const int GPUs[] = {0,5}; // If left blank all available GPUs will be used.

vector<int> g(GPUs, GPUs + sizeof(GPUs)/sizeof(int));

void configure(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
               vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
               hipStream_t stream[])
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipMalloc(&buffer_s[i], size);
        hipMalloc(&buffer_d[i], size);
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
        hipStreamCreate(&stream[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceEnablePeerAccess(g[j], 0);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceEnablePeerAccess(g[i], 0);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void reset(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
               vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
               hipStream_t stream[])
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipFree(buffer_s[i]);
        hipFree(buffer_d[i]);
        hipEventDestroy(start[i]);
        hipEventDestroy(stop[i]);
        hipStreamDestroy(stream[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceDisablePeerAccess(g[j]);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceDisablePeerAccess(g[i]);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void cudaMemcpyPoolAsync(
        int* &dst, int  dstDevice, int* &src, int  srcDevice,
        size_t count, int route, size_t chunk)
{
    void* rbuff[2];
    hipStream_t rstream[2];
    hipSetDevice(srcDevice);
    hipDeviceEnablePeerAccess(route, 0);
    hipStreamCreate(&rstream[0]);
    hipStreamCreate(&rstream[1]);
    hipSetDevice(dstDevice);
    hipStreamCreate(&rstream[0]);
    hipStreamCreate(&rstream[1]);
    hipSetDevice(route);
    hipDeviceEnablePeerAccess(dstDevice, 0);
    hipMalloc(&rbuff[0], chunk);
    hipMalloc(&rbuff[1], chunk);
    hipStreamCreate(&rstream[0]);
    hipStreamCreate(&rstream[1]);
    int strm=1;

    for(int i=0; i<count; i+=chunk)
    {
        strm^=(0^1);
        hipMemcpyPeerAsync(rbuff[strm], route, &src[i], srcDevice, chunk, rstream[strm]);
        hipMemcpyPeerAsync(&dst[i], dstDevice, rbuff[strm], route, chunk, rstream[strm]);
    }
    hipSetDevice(srcDevice);
    hipDeviceDisablePeerAccess(route);
    hipStreamDestroy(rstream[0]);
    hipStreamDestroy(rstream[1]);
    hipSetDevice(dstDevice);
    hipStreamDestroy(rstream[0]);
    hipStreamDestroy(rstream[1]);
    hipSetDevice(route);
    hipStreamDestroy(rstream[0]);
    hipStreamDestroy(rstream[1]);
    hipDeviceDisablePeerAccess(dstDevice);
    hipFree(rbuff[0]);
    hipFree(rbuff[1]);
}

void blocked_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
                  vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
                  hipStream_t stream[])
{
    float time_taken[g.size()*g.size()], bw[g.size()*g.size()];
    printf("\nBlocked Memory Transfers: Only one memory transfer at a time\n");

    configure(size, buffer_s, buffer_d, start, stop, stream);
    for (int i=0; i<g.size(); i++)
    {
        for (int j=0; j<g.size(); j++)
        {
            if (i!=j)
            {
                printf("Copying from %d to %d\n", g[i], g[j]);
                hipEventRecord(start[i]);
                //cudaMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[j],g[j], size);
                //cudaMemcpyPoolAsync(buffer_s[i],g[i],buffer_d[j],g[j], size, 1,
                        //size);
                hipEventRecord(stop[i]);
                hipEventSynchronize(stop[i]);
                hipDeviceSynchronize();
                float time_ms;
                hipEventElapsedTime(&time_ms,start[i],stop[i]);
                time_taken[i*g.size()+j] = time_ms*1e3;
                bw[i*g.size()+j] = (float)size*1000/time_ms/(1<<30);
            }
        }
    }

    printf("Time(ms) spent in memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        {
            if (i==j)
                printf("%12.2f", 0.0);
            else
                printf("%12.2f", time_taken[i*g.size()+j]);
        }
        printf("\n");
    }

    printf("bandwidth(Gbps) utilized during memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        if (i==j)
            printf("%12.2f", 0.0);
        else
            printf("%12.2f", bw[i*g.size()+j]);
        printf("\n");
    }
}

void perf_analyze(size_t size)
{
    vector<int*> buffer_s(g.size());
    vector<int*> buffer_d(g.size());
    vector<hipEvent_t> start(g.size());
    vector<hipEvent_t> stop(g.size());
    hipStream_t stream[g.size()];

    configure(size, buffer_s, buffer_d, start, stop, stream);

    // Cyclic
    blocked_copy(size, buffer_s, buffer_d, start, stop, stream);

    reset(size, buffer_s, buffer_d, start, stop, stream);
}

int main(int argc, char** argv)
{
    // NVLink D<->D performance
    size_t size = (1<<30);
    if (!g.size())
    {
        int n;
        printf("Using all 8 GPUs\n");
        hipGetDeviceCount(&n);
        for (int i=0; i<n; i++)
            g.push_back(i);
    }
    //define size
    perf_analyze(size);

    return 0;
}
