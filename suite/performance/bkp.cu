#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

using namespace std;

const int GPUs[] = {0,1,2}; // If left blank all available GPUs will be used.

vector<int> g(GPUs, GPUs + sizeof(GPUs)/sizeof(int));

void configure(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
               vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
               hipStream_t stream[])
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            hipMalloc(&buffer_s[i], size);
            hipMalloc(&buffer_d[i], size);
            hipEventCreate(&start[i]);
            hipEventCreate(&stop[i]);
            hipStreamCreate(&stream[i*g.size()+j]);
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceEnablePeerAccess(g[j], 0);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceEnablePeerAccess(g[i], 0);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void blocked_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
                  vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
                  hipStream_t stream[])
{
    float time_taken[g.size()*g.size()], bw[g.size()*g.size()];
    printf("\nBlocked Memory Transfers: Only one memory transfer at a time\n");

    configure(size, buffer_s, buffer_d, start, stop, stream);
    for (int i=0; i<g.size(); i++)
    {
        for (int j=0; j<g.size(); j++)
        {
            if (i!=j)
            {
                printf("Copying from %d to %d\n", g[i], g[j]);
                hipEventRecord(start[i]);
                hipMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[j],g[j], size);
                hipEventRecord(stop[i]);
                hipEventSynchronize(stop[i]);
                hipDeviceSynchronize();
                float time_ms;
                hipEventElapsedTime(&time_ms,start[i],stop[i]);
                time_taken[i*g.size()+j] = time_ms*1e3;
                bw[i*g.size()+j] = (float)size*1000/time_ms/(1<<30);
            }
        }
    }

    printf("Time(ms) spent in memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        {
            if (i==j)
                printf("%12.2f", 0.0);
            else
                printf("%12.2f", time_taken[i*g.size()+j]);
        }
        printf("\n");
    }

    printf("bandwidth(Gbps) utilized during memcpy\n");
    printf("   D\\D");
    for (int j=0; j<g.size(); j++)
        printf("%10d ", g[j]);

    printf("\n");

    for (int i=0; i<g.size(); i++)
    {
        printf("%6d", g[i]);
        for (int j=0; j<g.size(); j++)
        if (i==j)
            printf("%12.2f", 0.0);
        else
            printf("%12.2f", bw[i*g.size()+j]);
        printf("\n");
    }
}

void cyclic_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
                 vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
                 hipStream_t stream[])
{
    float bw[g.size()], time_taken[g.size()];
    printf("\nCyclic Memory Transfers: 0->1->2->3...n->0\n");
    
    configure(size, buffer_s, buffer_d, start, stop, stream);
    for (int i=0; i<g.size(); i++)
    {
        hipEventRecord(start[i], stream[i]);
        hipMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[(i+1)%g.size()],
                            g[(i+1)%g.size()], size, stream[i]);
        hipEventRecord(stop[i], stream[i]);
    }

    for (int i=0; i<g.size(); i++)
    {
        hipEventSynchronize(stop[i]);
        float time_ms;
        hipEventElapsedTime(&time_ms,start[i],stop[i]);
        time_taken[i] = time_ms*1e3;
        bw[i] = (float)size*1000/time_ms/(1<<30);
    }

    printf("\nTime spent in memcpy\n");
    for (int i=0; i<g.size(); i++)
        printf("GPU%d -> GPU%d:   %3.5f\n", g[i], g[(i+1)%g.size()], time_taken[i]);

    printf("\nBandwidth(Gbps) utilized in memcpy\n");
    for (int i=0; i<g.size(); i++)
        printf("GPU%d -> GPU%d:   %3.5f\n", g[i], g[(i+1)%g.size()], bw[i]);

}

void burst_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
                vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
                hipStream_t stream[])
{
    float bw[g.size()], time_taken[g.size()];
    printf("\nBurst copy: Every GPU is memcpy-ing to every other GPU\n");
    printf("%4d%8d%12s\n%4s%8s%12s\n%4s%8s%12s\n",
            1, 2,"n","^","^", "^","|","|","|");
    printf("3<-0->2 4<-1->3 ... %s<-%s->%s\n","q", "m", "p");
    printf("%4s%8s%12s\n%4s%8s%12s\n%4d%8d%12s\n\n",
            "|","|","|","v","v", "v",4,0,"r");
    configure(size, buffer_s, buffer_d, start, stop, stream);
    
    for (int i=0; i<g.size(); i++)
    {
        hipEventRecord(start[i]);
        for (int j=0; j<g.size(); j++)
            if (i!=j)
                hipMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[j],g[j], size,
                                    stream[i*g.size()+j]);
        hipEventRecord(stop[i]);
    }

    for (int i=0; i<g.size(); i++)
    {
        hipEventSynchronize(stop[i]);
        float time_ms;
        hipEventElapsedTime(&time_ms,start[i],stop[i]);
        time_taken[i] = time_ms*1e3;
        bw[i] = (float)size*1000/time_ms/(1<<30);
    }
    printf("\t\tTime(ms)\tBandwidth(Gbps)\n");
    for (int i=0; i<g.size(); i++)
        printf("GPU%d\t\t%6.2f\t\t%6.2f\n",g[i], time_taken[i], bw[i]);
}

void perf_analyze(size_t size)
{
    vector<int*> buffer_s(g.size());
    vector<int*> buffer_d(g.size());
    vector<hipEvent_t> start(g.size());
    vector<hipEvent_t> stop(g.size());
    hipStream_t stream[g.size() * g.size()];

    configure(size, buffer_s, buffer_d, start, stop, stream);

    // Blocked
    blocked_copy(size, buffer_s, buffer_d, start, stop, stream);

    // Cyclic
    cyclic_copy(size, buffer_s, buffer_d, start, stop, stream);

    // Burst
    burst_copy(size, buffer_s, buffer_d, start, stop, stream);
}

int main(int argc, char** argv)
{
    // NVLink D<->D performance
    size_t size = (1<<30);
    if (!g.size())
    {
        int n;
        printf("Using all 8 GPUs\n");
        hipGetDeviceCount(&n);
        for (int i=0; i<n; i++)
            g.push_back(i);
    }
    //define size
    perf_analyze(size);

    return 0;
}
