#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <vector>

using namespace std;

const int GPUs[] = {0,1,2,3,4}; // If left blank all available GPUs will be used.

vector<int> g(GPUs, GPUs + sizeof(GPUs)/sizeof(int));

void configure(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
               vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
               hipStream_t stream[])
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipMalloc(&buffer_s[i], size);
        hipMalloc(&buffer_d[i], size);
        hipEventCreate(&start[i]);
        hipEventCreate(&stop[i]);
        hipStreamCreate(&stream[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceEnablePeerAccess(g[j], 0);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceEnablePeerAccess(g[i], 0);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void reset(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
               vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
               hipStream_t stream[])
{
    for (int i=0; i<g.size(); i++)
    {
        hipSetDevice(g[i]);
        hipFree(buffer_s[i]);
        hipFree(buffer_d[i]);
        hipEventDestroy(start[i]);
        hipEventDestroy(stop[i]);
        hipStreamDestroy(stream[i]);
        for (int j=0; j<g.size(); j++)
        {
            int access;
            if (i!=j)
            {
                hipDeviceCanAccessPeer(&access, g[i], g[j]);
                if (access)
                {
                    hipSetDevice(g[i]);
                    hipDeviceDisablePeerAccess(g[j]);
                    hipDeviceSynchronize();
                    hipSetDevice(g[j]);
                    hipDeviceDisablePeerAccess(g[i]);
                    hipDeviceSynchronize();
                }
            }
        }
    }
}

void cyclic_copy(size_t size, vector<int*> &buffer_s, vector<int*> &buffer_d,
                 vector<hipEvent_t> &start, vector<hipEvent_t> &stop,
                 hipStream_t stream[])
{
    float bw[g.size()], time_taken[g.size()];
    printf("\nCyclic Memory Transfers: 0->1->2->3...n->0\n");
    
    configure(size, buffer_s, buffer_d, start, stop, stream);
    for (int i=0; i<g.size(); i++)
    {
        hipEventRecord(start[i], stream[i]);
        hipMemcpyPeerAsync(buffer_s[i],g[i],buffer_d[(i+1)%g.size()],
                            g[(i+1)%g.size()], size, stream[i]);
        hipEventRecord(stop[i], stream[i]);
    }

    for (int i=0; i<g.size(); i++)
    {
        hipEventSynchronize(stop[i]);
        float time_ms;
        hipEventElapsedTime(&time_ms,start[i],stop[i]);
        time_taken[i] = time_ms*1e3;
        bw[i] = (float)size*1000/time_ms/(1<<30);
    }

    printf("\nTime(s) spent in memcpy\n");
    for (int i=0; i<g.size(); i++)
        printf("GPU%d -> GPU%d:   %3.5f\n", g[i], g[(i+1)%g.size()],
                time_taken[i]/1e3);

    printf("\nBandwidth(Gbps) utilized in memcpy\n");
    for (int i=0; i<g.size(); i++)
        printf("GPU%d -> GPU%d:   %3.5f\n", g[i], g[(i+1)%g.size()], bw[i]);

}

void perf_analyze(size_t size)
{
    vector<int*> buffer_s(g.size());
    vector<int*> buffer_d(g.size());
    vector<hipEvent_t> start(g.size());
    vector<hipEvent_t> stop(g.size());
    hipStream_t stream[g.size()];

    configure(size, buffer_s, buffer_d, start, stop, stream);

    // Cyclic
    cyclic_copy(size, buffer_s, buffer_d, start, stop, stream);

    reset(size, buffer_s, buffer_d, start, stop, stream);
}

int main(int argc, char** argv)
{
    // NVLink D<->D performance
    size_t size = (1<<30);
    if (!g.size())
    {
        int n;
        printf("Using all 8 GPUs\n");
        hipGetDeviceCount(&n);
        for (int i=0; i<n; i++)
            g.push_back(i);
    }
    //define size
    perf_analyze(size);

    return 0;
}
