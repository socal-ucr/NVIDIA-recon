
#include <hip/hip_runtime.h>
#include <stdio.h>

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}



const int N = 2048;

// CUDA Kernel for Vector Addition
__global__ void Vector_Addition ( const int *dev_a , const int *dev_b , int *dev_c)
{
      //Get the id of thread within a block
      unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x ;
     

      while ( tid < N ) // check the boundry condition for the threads
      {          
            dev_c [tid] = dev_a[tid] + dev_b[tid] ;

            tid+= blockDim.x * gridDim.x ;
      }
}

int main (void)
{

      //Host array
      int Host_a[N], Host_b[N], Host_c[N];

      //Device array
      int *dev_a , *dev_b, *dev_c ;

      //Allocate the memory on the GPU
      HANDLE_ERROR ( hipMalloc((void **)&dev_a , N*sizeof(int) ) );
      HANDLE_ERROR ( hipMalloc((void **)&dev_b , N*sizeof(int) ) );
      HANDLE_ERROR ( hipMalloc((void **)&dev_c , N*sizeof(int) ) );

      //fill the Host array with random elements on the CPU
      for ( int i = 0; i <N ; i++ )
      {
            Host_a[i] = -i ;
            Host_b[i] = i*i ; 
      }

      //Copy Host array to Device array
      HANDLE_ERROR (hipMemcpy (dev_a , Host_a , N*sizeof(int) , hipMemcpyHostToDevice));
      HANDLE_ERROR (hipMemcpy (dev_b , Host_b , N*sizeof(int) , hipMemcpyHostToDevice));

      //Make a call to GPU kernel
      Vector_Addition <<< (N+127)/128, 128  >>> (dev_a , dev_b , dev_c ) ;

      //Copy back to Host array from Device array
      HANDLE_ERROR (hipMemcpy(Host_c , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost));

      //Display the result
      for ( int i = 0; i<N; i++ )
                  printf ("%d + %d = %d\n", Host_a[i] , Host_b[i] , Host_c[i] ) ;

      //Free the Device array memory
      hipFree (dev_a) ;
      hipFree (dev_b) ;
      hipFree (dev_c) ;

      system("pause");
      return 0 ;

}
